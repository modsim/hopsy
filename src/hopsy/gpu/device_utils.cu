#include "device_utils.h"
#include <hip/hip_runtime.h>
#include "helper.h"



namespace hopsy {
    namespace GPU {

        void set_device(int device) {
            CUDA_CHECK(hipSetDevice(device));
        }

        std::vector<std::string> list_devices() {
            int count;
            CUDA_CHECK(hipGetDeviceCount(&count));
            std::vector<std::string> devices;
            for (int i = 0; i < count; ++i) {
                hipDeviceProp_t prop;
                CUDA_CHECK(hipGetDeviceProperties(&prop, i));
                std::ostringstream oss;
                oss << "[" << i << "] " << prop.name;
                devices.push_back(oss.str());
            }
            return devices;
        }
    }
}
